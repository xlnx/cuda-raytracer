#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <utility>
#include <queue>
#include <iostream>
#include <util/exception.hpp>
#include <vec/vmath.hpp>
#include "mesh.hpp"

#define KOISHI_TRIANGLE_STRIPE 32

namespace koishi
{
namespace core
{
struct TriangleInfo
{
	uint3 index;
	double3 vmax;
	double3 vmin;
	double area;
};

static BVHTree createBVH( std::vector<TriangleInfo> &info )
{
	struct QueueItem
	{
		uint index;
		std::vector<TriangleInfo>::iterator begin, end;
	};
	BVHTree::buffer_type res( 2 );
	std::queue<QueueItem> Q;
	Q.emplace( QueueItem{ 1, info.begin(), info.end() } );
	while ( !Q.empty() )
	{
		uint index = Q.front().index;
		auto begin = Q.front().begin;
		auto end = Q.front().end;
		Q.pop();

		BVHNode node;  // current bbox
		node.vmax = begin->vmax;
		node.vmin = begin->vmin;
		double s = 0;
		for ( auto iter = begin; iter != end; ++iter )
		{
			node.vmax = max( node.vmax, iter->vmax );
			node.vmin = min( node.vmin, iter->vmin );
			s += iter->area;
		}
		node.begin = ( begin - info.begin() ) * 3;
		node.end = ( end - info.begin() ) * 3;
		node.isleaf = end - begin <= KOISHI_TRIANGLE_STRIPE;
		if ( index >= res.size() )
		{
			res.resize( index + 1 );
		}
		res[ index ] = node;
		if ( !node.isleaf )
		{
			s /= 2;
			auto w = node.vmax - node.vmin;
			if ( w.x >= w.y && w.x >= w.z )
			{
				std::sort( begin, end, []( const TriangleInfo &a, const TriangleInfo &b ) { return a.vmin.x < b.vmin.x; } );
			}
			else if ( w.y >= w.x && w.y >= w.z )
			{
				std::sort( begin, end, []( const TriangleInfo &a, const TriangleInfo &b ) { return a.vmin.y < b.vmin.y; } );
			}
			else
			{
				std::sort( begin, end, []( const TriangleInfo &a, const TriangleInfo &b ) { return a.vmin.z < b.vmin.z; } );
			}
			for ( auto iter = begin; iter != end; ++iter )
			{
				if ( ( s -= iter->area ) <= 0.f )
				{
					Q.emplace( QueueItem{ index << 1, begin, iter } );
					Q.emplace( QueueItem{ ( index << 1 ) + 1, iter, end } );
					break;
				}
			}
		}
	}
	return std::move( res );
}

static void printBVH( const BVHTree &tr, uint index = 1 )
{
	std::cout << index << " ";
	if ( !tr[ index ].isleaf )
	{
		printBVH( tr, index << 1 );
		printBVH( tr, ( index << 1 ) + 1 );
	}
}

KOISHI_HOST_DEVICE bool Mesh::intersect( const Ray &ray, uint root, Hit &hit ) const
{
	uint i = root;
	while ( !bvh[ i ].isleaf )
	{
		auto left = ray.intersect_bbox( bvh[ i << 1 ].vmin, bvh[ i << 1 ].vmax );
		auto right = ray.intersect_bbox( bvh[ ( i << 1 ) + 1 ].vmin, bvh[ ( i << 1 ) + 1 ].vmax );
		if ( !left && !right ) return false;
		if ( left && right )
		{
			Hit hit1;
			auto b0 = intersect( ray, root << 1, hit );
			auto b1 = intersect( ray, ( root << 1 ) | 1, hit1 );
			if ( !b0 && !b1 )
			{
				return false;
			}
			if ( !b0 || b1 && hit1.t < hit.t )
			{
				hit = hit1;
			}
			return true;
		}
		i <<= 1;
		if ( right ) i |= 1;
	}
	// return true;
	hit.t = INFINITY;
	for ( uint j = bvh[ i ].begin; j < bvh[ i ].end; j += 3 )
	{
		Hit hit1;
		if ( ray.intersect_triangle( vertices[ indices[ j ] ],
									 vertices[ indices[ j + 1 ] ],
									 vertices[ indices[ j + 2 ] ], hit1 ) &&
			 hit1.t < hit.t )
		{
			hit = hit1;
			hit.id = j;
		}
	}
	return hit.t != INFINITY;
}

void PolyMesh::collectObjects( const aiScene *scene, const aiNode *node, const aiMatrix4x4 &tr )
{
	auto trans = tr * node->mTransformation;
	for ( uint i = 0; i != node->mNumMeshes; ++i )
	{
		auto aimesh = scene->mMeshes[ node->mMeshes[ i ] ];
		PolyVector<double3> vertices;
		if ( aimesh->HasPositions() )
		{
			vertices.resize( aimesh->mNumVertices );
			for ( uint j = 0; j != aimesh->mNumVertices; ++j )
			{
				auto v = trans * aimesh->mVertices[ j ];
				vertices[ j ] = double3{ v.x, v.y, v.z };
			}
		}
		PolyVector<double3> normals;
		if ( aimesh->HasNormals() )
		{
			normals.resize( aimesh->mNumVertices );
			for ( uint j = 0; j != aimesh->mNumVertices; ++j )
			{
				normals[ j ] = double3{ aimesh->mNormals[ j ].x,
										aimesh->mNormals[ j ].y,
										aimesh->mNormals[ j ].z };
			}
		}
		std::vector<TriangleInfo> indices;
		if ( aimesh->HasFaces() )
		{
			for ( uint j = 0; j != aimesh->mNumFaces; ++j )
			{
				if ( aimesh->mFaces[ j ].mNumIndices == 3 )
				{
					auto index = uint3{ aimesh->mFaces[ j ].mIndices[ 0 ],
										aimesh->mFaces[ j ].mIndices[ 1 ],
										aimesh->mFaces[ j ].mIndices[ 2 ] };
					double3 v[] = { vertices[ index.x ], vertices[ index.y ], vertices[ index.z ] };
					TriangleInfo info;
					info.index = index;
					info.vmax = max( v[ 0 ], max( v[ 1 ], v[ 2 ] ) );
					info.vmin = min( v[ 0 ], min( v[ 1 ], v[ 2 ] ) );
					info.area = length( cross( v[ 2 ] - v[ 0 ], v[ 1 ] - v[ 0 ] ) );
					indices.emplace_back( std::move( info ) );
				}
			}
		}
		if ( indices.size() <= 0 ) continue;

		Mesh m;
		// m.emissive = default_config.emissive;
		// m.color = default_config.color;
		m.bvh = createBVH( indices );
		std::cout << "Successfully buit BVH: " << m.bvh.size() << std::endl;
		m.vertices = std::move( vertices );
		m.normals = std::move( normals );
		m.matid = aimesh->mMaterialIndex;
		PolyVector<uint> idxBuffer( indices.size() * 3 );
		for ( uint j = 0; j != indices.size(); ++j )
		{
			idxBuffer[ 3 * j ] = indices[ j ].index.x;
			idxBuffer[ 3 * j + 1 ] = indices[ j ].index.y;
			idxBuffer[ 3 * j + 2 ] = indices[ j ].index.z;
		}
		m.indices = std::move( idxBuffer );
		mesh.emplace_back( std::move( m ) );
	}
	for ( auto i = 0u; i != node->mNumChildren; ++i )
	{
		collectObjects( scene, node->mChildren[ i ], trans );
	}
}

PolyMesh::PolyMesh( PolyVector<double3> &&vertices,
					PolyVector<double3> &&normals,
					const std::vector<uint3> &idx )
{
	std::vector<TriangleInfo> indices;
	for ( auto &index : idx )
	{
		double3 v[] = { vertices[ index.x ], vertices[ index.y ], vertices[ index.z ] };
		TriangleInfo info;
		info.index = index;
		info.vmax = max( v[ 0 ], max( v[ 1 ], v[ 2 ] ) );
		info.vmin = min( v[ 0 ], min( v[ 1 ], v[ 2 ] ) );
		info.area = length( cross( v[ 2 ] - v[ 0 ], v[ 1 ] - v[ 0 ] ) );
		indices.emplace_back( std::move( info ) );
	}
	Mesh m;
	m.bvh = createBVH( indices );
	std::cout << "Successfully buit BVH: " << m.bvh.size() << std::endl;
	m.vertices = std::move( vertices );
	m.normals = std::move( normals );
	PolyVector<uint> idxBuffer( indices.size() * 3 );
	for ( uint j = 0; j != indices.size(); ++j )
	{
		idxBuffer[ 3 * j ] = indices[ j ].index.x;
		idxBuffer[ 3 * j + 1 ] = indices[ j ].index.y;
		idxBuffer[ 3 * j + 2 ] = indices[ j ].index.z;
	}
	m.indices = std::move( idxBuffer );
	mesh.emplace_back( std::move( m ) );
}

PolyMesh::PolyMesh( const aiScene *scene )
{
	auto t = scene->mRootNode->mTransformation;
	collectObjects( scene, scene->mRootNode, t );
	for ( uint i = 0; i != scene->mNumMaterials; ++i )
	{
		auto mat = scene->mMaterials[ i ];
		aiString name;
		mat->Get( AI_MATKEY_NAME, name );
		material.emplace_back( name.C_Str() );
	}
}

}  // namespace core

}  // namespace koishi
