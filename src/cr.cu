#include "hip/hip_runtime.h"
#include <sstream>
#include <core/tracer.hpp>
#include <core/random.hpp>
#include <core/radiance.hpp>
#include <core/renderer.hpp>
#include <vis/renderer.hpp>

using namespace koishi;
using namespace core;

#if 1
struct PolyStruct( A )
{
	Poly( int i ) :
	  n( i )
	{
		PolyVector<int> vv;
		for ( int i = 0; i != n; ++i )
		{
			vv.emplace_back( i );
		}
		v = std::move( vv );
	}
	KOISHI_HOST_DEVICE Poly(Poly &&other) = default;
	Poly(const Poly &other):
	  n( other.n ),
	  v( other.v )
	{
		std::cout << other.n << " " << other.v.size() << " "<< other.v.data() << std::endl;
		std::cout << n << " " <<  v.size() << " " << v.data() << std::endl;
	}

	__host__ __device__ virtual int f()
	{
		int s = 0;
		for ( int i = 0; i != v.size(); ++i )
		{
		//	s += v[ i ];
		}
		return s;
	}

	int n;
	PolyVectorView<int> v;
};

__global__ void add( PolyVectorView<A> vec, PolyVectorView<int*> res, PolyVectorView<int> n )
{
	res[0] = vec[1].v.data();
	n[0] = vec[1].v.size();
}
#endif

int main( int argc, char **argv )
{
#if 1
	PolyVector<A> vec;
	for ( int i = 0; i != 2; ++i )
	{
		vec.emplace_back( i );
	}
	PolyVectorView<A> view = std::move( vec );
	view.emitAndReplace();
	PolyVectorView<int*> res( 3 );
	PolyVectorView<int> nn( 3 );
	res.emitAndReplace();
	nn.emitAndReplace();
	add<<<1, 1>>>( view.forward(), res.forward(), nn.forward() );
	hipDeviceSynchronize();
	res.fetchAndReplace();
	nn.fetchAndReplace();
	std::cout << res[ 0 ] << " " << nn[0] << std::endl;
	return 0;
#endif

	if ( std::string( argv[ 2 ] ) == "-v" )
	{
	}
	else
	{
		uint spp;
		std::istringstream is( argv[ 3 ] );
		is >> spp;

		using TraceFn = Radiance<DRand48>;
		Renderer<Tracer<TraceFn>> r{ 1024, 768 };

		r.render( argv[ 1 ], argv[ 2 ], spp );
	}
}
