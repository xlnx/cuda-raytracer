#include <cstdio>
#include <core/kernel/radiance.hpp>
#include <core/kernel/normal.hpp>
#include <core/kernel/custom.hpp>
// #include <core/kernel/bruteForce.hpp>
#include <core/tracer/cpuMulticore.hpp>
//#include <core/tracer/cudaSingleGPU.hpp>
// #include <core/renderer/factory.hpp>
#include <core/renderer/renderer.hpp>
#include <vis/renderer.hpp>
#include <cxxopts/cxxopts.hpp>

using namespace koishi;
using namespace core;

int main( int argc, char **argv )
{
	cxxopts::Options options( "cr", "Ray tracer for heterogeneous systems, by KoishiChan~" );
	options.add_options()(
	  "v,visualize", "Visualize BVH using openGL." )(
	  "o", "Place the output into <file>.", cxxopts::value<std::string>()->default_value( "a.png" ) )(
	  "s,sample-per-pixel", "Number of sample points per pixel.", cxxopts::value<uint>()->default_value( "1" ) )(
	  "l,list", "List all valid renderers." )(
	  "h,help", "Show help message." )(
	  "t,tracer", "Specify target tracer.", cxxopts::value<std::string>()->default_value( "CPUMultiCoreTracer" ) )(
	  "k,kernel", "Specify kernel function.", cxxopts::value<std::string>()->default_value( "Radiance" ) )(
	  "a,allocator", "Specify allocator.", cxxopts::value<std::string>()->default_value( "HybridAllocator" ) )(
	  "resolution", "Specify target resolution.", cxxopts::value<std::string>()->default_value( "1024x768" ) );

	try
	{
		auto opt = options.parse( argc, argv );

		if ( opt.count( "h" ) )
		{
			KLOG( options.help() );
		}
		else if ( opt.count( "v" ) )
		{
#ifdef KOISHI_USE_GL
			vis::Renderer r{ 1024, 768 };
			r.render( argv[ 1 ] );
#endif
		}
		else
		{
			if ( opt.count( "l" ) )
			{
				// for ( auto &e : factory.getValidTypes() )
				// {
				// 	KLOG( e );
				// }
			}
			else
			{
				uint w = 1024, h = 768;
				auto resolution = opt[ "resolution" ].as<std::string>();
				sscanf( resolution.c_str(), "%ux%u", &w, &h );

				Renderer renderer( w, h );

				auto spp = opt[ "s" ].as<uint>();
				auto out = opt[ "o" ].as<std::string>();

				KLOG( "Sample", spp, "points per pixel" );

				renderer.render( argv[ 1 ], out, spp );
			}
		}
	}
	catch ( const std::exception &err )
	{
		KINFO( fatal, err.what() );
	}
}
