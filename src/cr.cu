#include <sstream>
#include <core/random.hpp>
#include <core/radiance.hpp>
#include <core/renderer.hpp>
#include <core/tracer/cpuMulticore.hpp>
#include <core/tracer/cudaSingleGPU.hpp>
#include <core/factory.hpp>
#include <vis/renderer.hpp>

using namespace koishi;
using namespace core;

int main( int argc, char **argv )
{
	if ( std::string( argv[ 2 ] ) == "-v" )
	{
#ifdef KOISHI_USE_GL
		vis::Renderer r{ 1024, 768 };
		r.render( argv[ 1 ] );
#endif
	}
	else
	{
		uint spp;
		std::istringstream is( argv[ 3 ] );
		is >> spp;

		Factory<
		  templates<CPUMultiCoreTracer
#ifdef KOISHI_USE_CUDA
					  CudaSingleGPUTracer
#endif
					>,
		  templates<Radiance>,
		  types<FakeRand, DRand48>>
		  factory;

		auto r = factory.create( "CPUMultiCoreTracer", 1024, 768 );

		r->render( argv[ 1 ], argv[ 2 ], spp );

		// 		using TraceFn = Radiance<FakeRand>;
		// 		Renderer<
		// #ifdef KOISHI_USE_CUDA
		// 		  cuda::
		// #endif
		// 			Tracer<
		// 			  TraceFn
		// #ifndef KOISHI_USE_CUDA
		// 		//   ,
		// 		//   HybridAllocator
		// 		//   1
		// #endif
		// 			  >>
		// 		  r{ 1024, 768 };
	}
}
