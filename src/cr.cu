#include "hip/hip_runtime.h"
#include <sstream>
#include <core/tracer.hpp>
#include <core/random.hpp>
#include <core/radiance.hpp>
#include <core/renderer.hpp>
#include <vis/renderer.hpp>

using namespace koishi;
using namespace core;

#if 1
struct PolyStruct( A )
{
	Poly( int i ) :
	  n( i )
	{
		PolyVector<int> vv;
		for ( int i = 0; i <= n; ++i )
		{
			vv.emplace_back( i );
		}
		v = std::move( vv );
	}

	__host__ __device__ virtual int f() const
	{
		int s = v.size() * 1000;
		for ( int i = 0; i != v.size(); ++i )
		{
			s += v[ i ];
		}
		return s;
	}

	int n;
	PolyVectorView<int> v;
};

__global__ void add( const PolyVectorView<A> &vec, PolyVectorView<int> &n, PolyVectorView<const int*> &p )
{
	//n[0] = 1; n[1] = 2;
	//n[0] = 1;
	for (auto i = 0; i != vec.size(); ++i)
		//n[i] = 1;
		n[i] = vec[i].f(), p[i] = vec[i].v.data();
}
#endif

int main( int argc, char **argv )
{
#if 1
	PolyVector<A> vec;
	for ( int i = 0; i != 10; ++i )
	{
		vec.emplace_back( i );
	}
	PolyVectorView<A> view = std::move( vec );

	LOG( view.size(), view.data() );
	
	view.emitAndReplace();
	
	LOG( view.size(), view.data() );
	
	PolyVectorView<int> nn( view.size() );
	PolyVectorView<const int*> pp( view.size() );
	
	LOG(nn.space(), nn.data() );
	LOG(pp.space(), pp.data() );
	
	LOG("size of nn", nn.size());

	nn.emitAndReplace();
	pp.emitAndReplace();

	LOG(nn.space(), nn.data());
	LOG(pp.space(), pp.data());
	
	LOG("size of nn", nn.size());

	kernel(add, 1, 1)(view, nn, pp);
	
	LOG(nn.space(), nn.data());
	LOG(pp.space(), pp.data());

	LOG("size of nn", nn.size());
	
	nn.fetchAndReplace();
	pp.fetchAndReplace();
	
	LOG(nn.space(), nn.data());
	LOG(pp.space(), pp.data());

	LOG("size of nn", nn.size());

	for ( auto &e: nn )
		std::cout << e << std::endl;
	
	LOG( "normal exit" );

	return 0;
#endif

	if ( std::string( argv[ 2 ] ) == "-v" )
	{
	}
	else
	{
		uint spp;
		std::istringstream is( argv[ 3 ] );
		is >> spp;

		using TraceFn = Radiance<DRand48>;
		Renderer<Tracer<TraceFn>> r{ 1024, 768 };

		r.render( argv[ 1 ], argv[ 2 ], spp );
	}
}
