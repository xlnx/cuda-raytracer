#include "hip/hip_runtime.h"
#include <sstream>
#include <core/tracer.hpp>
#include <core/random.hpp>
#include <core/radiance.hpp>
#include <core/renderer.hpp>
#include <vis/renderer.hpp>

using namespace koishi;
using namespace core;

#if 1
struct PolyStruct( A )
{
	Poly( int i ) :
	  n( i )
	{
		PolyVector<int> vv;
		for ( int i = 0; i != n; ++i )
		{
			vv.emplace_back( i );
		}
		v = std::move( vv );
	}
	__host__ __device__ virtual int f()
	{
		int s = 0;
		for ( int i = 0; i != v.size(); ++i )
		{
			s += v[ i ];
		}
		return s;
	}

private:
	int n;
	PolyVectorView<int> v;
};

__global__ void add( PolyVectorView<A> vec, PolyVectorView<int> res )
{
	res[ 0 ] = 0;
	for ( auto &e : vec )
		res[ 0 ] += e.f();
}
#endif

int main( int argc, char **argv )
{
#if 1
	PolyVector<A> vec;
	for ( int i = 0; i != 10; ++i )
	{
		vec.emplace_back( i );
	}
	PolyVectorView<A> view = std::move( vec );
	view.emitAndReplace();
	PolyVectorView<int> res( 3 );
	res.emitAndReplace();
	add<<<1, 1>>>( view.forward(), res.forward() );
	hipDeviceSynchronize();
	res.fetchAndReplace();
	std::cout << res[ 0 ] << std::endl;
	return 0;
#endif

	if ( std::string( argv[ 2 ] ) == "-v" )
	{
	}
	else
	{
		uint spp;
		std::istringstream is( argv[ 3 ] );
		is >> spp;

		using TraceFn = Radiance<DRand48>;
		Renderer<Tracer<TraceFn>> r{ 1024, 768 };

		r.render( argv[ 1 ], argv[ 2 ], spp );
	}
}
