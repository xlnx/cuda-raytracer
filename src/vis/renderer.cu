
#include <hip/hip_runtime.h>
#if defined( KOISHI_USE_GL )

#include <fstream>
#include "renderer.hpp"
#include "camera.hpp"

namespace koishi
{
namespace vis
{
Renderer::Renderer( uint w, uint h ) :
  w( w ), h( h )
{
	if ( !glfwInit() )
	{
		throw util::Exception( "Failed to init glfw." );
	}
	glfwWindowHint( GLFW_VERSION_MAJOR, 3 );
	glfwWindowHint( GLFW_VERSION_MINOR, 3 );
	glfwWindowHint( GLFW_SAMPLES, 4 );
	glfwWindowHint( GLFW_RESIZABLE, false );
	window = glfwCreateWindow( w, h, "Scene Preview", nullptr, nullptr );
	glfwMakeContextCurrent( window );
	if ( !gladLoadGLLoader( (GLADloadproc)glfwGetProcAddress ) )
	{
		throw util::Exception( "Failed to get GL proc address." );
	}
	glEnable( GL_DEPTH_TEST );
	glClearColor( 0, 0, 0, 0 );
}

Renderer::~Renderer()
{
	glfwTerminate();
}

static GLuint compileShader()
{
	auto vs = glCreateShader( GL_VERTEX_SHADER );
	auto fs = glCreateShader( GL_FRAGMENT_SHADER );
	const char *vsrc[] = {
#include "vis.vert"
	};
	const char *fsrc[] = {
#include "vis.frag"
	};
	glShaderSource( vs, 1, vsrc, nullptr );
	glShaderSource( fs, 1, fsrc, nullptr );
	GLint success;
	glCompileShader( vs );
	glGetShaderiv( vs, GL_COMPILE_STATUS, &success );
	assert( success );
	glCompileShader( fs );
	glGetShaderiv( fs, GL_COMPILE_STATUS, &success );
	assert( success );

	auto prog = glCreateProgram();
	glAttachShader( prog, vs );
	glAttachShader( prog, fs );
	glLinkProgram( prog );
	glGetProgramiv( prog, GL_LINK_STATUS, &success );
	assert( success );

	return prog;
}

struct SubMesh
{
	GLuint vao;
	core::BVHTree bvh;
};

void Renderer::render( const std::string &path )
{
	jsel::Scene scene;
	std::ifstream( path ) >> scene;
	if ( !scene.camera.size() )
	{
		throw util::Exception( "No valid camera in this scene." );
	}
	Camera camera( w, h, scene.camera[ 0 ] );
	std::vector<SubMesh> mesh;
	for ( auto &m : scene.mesh )
	{
		for ( auto &e : core::PolyMesh( m ).mesh )
		{
			GLuint vao, vbo, ebo;
			glGenVertexArrays( 1, &vao );
			glGenBuffers( 1, &vbo );
			glGenBuffers( 1, &ebo );
			glBindVertexArray( vao );
			glBindBuffer( GL_ARRAY_BUFFER, vbo );
			glBindBuffer( GL_ELEMENT_ARRAY_BUFFER, ebo );
			glBufferData( GL_ARRAY_BUFFER, e.vertices.size() * sizeof( e.vertices[ 0 ] ), &e.vertices[ 0 ], GL_STATIC_DRAW );
			glBufferData( GL_ELEMENT_ARRAY_BUFFER, e.indices.size() * sizeof( e.indices[ 0 ] ), &e.indices[ 0 ], GL_STATIC_DRAW );
			glEnableVertexAttribArray( 0 );
			glVertexAttribPointer( 0, 3, GL_DOUBLE, GL_FALSE, sizeof( double3 ), (const void *)( 0 ) );
			glBindVertexArray( 0 );
			glBindBuffer( GL_ARRAY_BUFFER, 0 );
			glBindBuffer( GL_ELEMENT_ARRAY_BUFFER, 0 );
			mesh.emplace_back( SubMesh{ vao, e.bvh } );
		}
	}

	auto prog = compileShader();
	glUseProgram( prog );

	while ( !glfwWindowShouldClose( window ) )
	{
		glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT );

		static auto prev = glfwGetTime();
		auto curr = glfwGetTime();
		auto detMillis = curr - prev;

		auto mat = camera.getTrans();
		glUniformMatrix4fv( glGetUniformLocation( prog, "wvp" ), 1, GL_FALSE,
							reinterpret_cast<const float *>( &mat ) );

		auto k = 5;
		for ( auto &m : mesh )
		{
			glPolygonMode( GL_FRONT_AND_BACK, GL_LINE );
			glUniform1i( glGetUniformLocation( prog, "mode" ), 0 );
			glBindVertexArray( m.vao );
			glDrawElements( GL_TRIANGLES, m.bvh[ 1 ].end - m.bvh[ 1 ].begin, GL_UNSIGNED_INT, nullptr );
			glBindVertexArray( 0 );

			glPolygonMode( GL_FRONT_AND_BACK, GL_FILL );
			glUniform1i( glGetUniformLocation( prog, "mode" ), 1 );
			glBindVertexArray( m.vao );
			glDrawElements( GL_TRIANGLES, m.bvh[ k ].end - m.bvh[ k ].begin,
							GL_UNSIGNED_INT, (uint *)nullptr + m.bvh[ k ].begin );
			glBindVertexArray( 0 );
		}

		glfwPollEvents();
		glfwSwapBuffers( window );
	}
}

}  // namespace vis

}  // namespace koishi

#endif
