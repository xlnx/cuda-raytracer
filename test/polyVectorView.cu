#include "hip/hip_runtime.h"
#include <sstream>
#include <core/tracer.hpp>
#include <core/random.hpp>
#include <core/radiance.hpp>
#include <core/renderer.hpp>
#include <vis/renderer.hpp>

#include <gtest/gtest.h>

using namespace koishi;
using namespace core;

#if KOISHI_USE_CUDA

struct PolyStruct( A )
{
	Poly( int i ) :
	  n( i )
	{
		PolyVector<int> vv;
		for ( int i = 0; i <= n; ++i )
		{
			vv.emplace_back( i );
		}
		v = std::move( vv );
	}

	__host__ __device__ virtual int f() const
	{
		int s = v.size() * 1000;
		for ( int i = 0; i != v.size(); ++i )
		{
			s += v[ i ];
		}
		return s;
	}

	int n;
	PolyVectorView<int> v;
};

__global__ void add( const PolyVectorView<A> &vec, PolyVectorView<int> &n, PolyVectorView<const int *> &p )
{
	//n[0] = 1; n[1] = 2;
	//n[0] = 1;
	for ( auto i = 0; i != vec.size(); ++i )
		//n[i] = 1;
		n[ i ] = vec[ i ].f(), p[ i ] = vec[ i ].v.data();
}

#endif

TEST( first_poly_vector_test_case, struct_with_non_standard_layout )
{
#if KOISHI_USE_CUDA
	PolyVector<A> vec;
	for ( int i = 0; i != 10; ++i )
	{
		vec.emplace_back( i );
	}
	PolyVectorView<A> view = std::move( vec );

	EXPECT_EQ( view.size(), 10 );
	LOG( view.data() );

	view.emitAndReplace();

	EXPECT_EQ( view.size(), 10 );
	LOG( view.data() );

	PolyVectorView<int> nn( view.size() );
	PolyVectorView<const int *> pp( view.size() );

	EXPECT_EQ( nn.space(), 0 );
	LOG( nn.data() );
	EXPECT_EQ( pp.space(), 0 );
	LOG( pp.data() );

	EXPECT_EQ( 10, nn.size() );

	nn.emitAndReplace();
	pp.emitAndReplace();

	EXPECT_EQ( nn.space(), 1 );
	LOG( nn.data() );
	EXPECT_EQ( pp.space(), 1 );
	LOG( pp.data() );

	EXPECT_EQ( 10, nn.size() );

	kernel( add, 1, 1 )( view, nn, pp );

	EXPECT_EQ( nn.space(), 1 );
	LOG( nn.data() );
	EXPECT_EQ( pp.space(), 1 );
	LOG( pp.data() );

	EXPECT_EQ( 10, nn.size() );

	nn.fetchAndReplace();
	pp.fetchAndReplace();

	EXPECT_EQ( nn.space(), 0 );
	LOG( nn.data() );
	EXPECT_EQ( pp.space(), 0 );
	LOG( pp.data() );

	EXPECT_EQ( 10, nn.size() );

	for ( auto &e : nn )
		std::cout << e << std::endl;
	
	int ss = 0;

	for (int i = 0; i != nn.size(); ++i)
	{
		ss += i;
		EXPECT_EQ( nn[i], ss + 1000 * ( i + 1 ) );
	}

	LOG( "normal exit" );
#else
	LOG( "no cuda toolkit provided" );

	EXPECT_EQ( 1, 1 );
#endif
}
