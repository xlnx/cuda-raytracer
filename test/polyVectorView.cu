#include "hip/hip_runtime.h"
#include <sstream>
#include <core/tracer.hpp>
#include <core/random.hpp>
#include <core/radiance.hpp>
#include <core/renderer.hpp>
#include <vis/renderer.hpp>

#include <gtest/gtest.h>

using namespace koishi;
using namespace core;

#if KOISHI_USE_CUDA

struct PolyStruct( A )
{
	Poly( int i ) :
	  n( i )
	{
		PolyVector<int> vv;
		for ( int i = 0; i <= n; ++i )
		{
			vv.emplace_back( i );
		}
		v = std::move( vv );
	}

	__host__ __device__ virtual int f() const
	{
		int s = v.size() * 1000;
		for ( int i = 0; i != v.size(); ++i )
		{
			s += v[ i ];
		}
		return s;
	}

	int n;
	PolyVectorView<int> v;
};

__global__ void add( const PolyVectorView<A> &vec, PolyVectorView<int> &n, PolyVectorView<const int *> &p )
{
	//n[0] = 1; n[1] = 2;
	//n[0] = 1;
	for ( auto i = 0; i != vec.size(); ++i )
		//n[i] = 1;
		n[ i ] = vec[ i ].f(), p[ i ] = vec[ i ].v.data();
}

#endif

TEST( first_poly_vector_test_case, struct_with_non_standard_layout )
{
#if KOISHI_USE_CUDA
	PolyVector<A> vec;
	for ( int i = 0; i != 10; ++i )
	{
		vec.emplace_back( i );
	}
	PolyVectorView<A> view = std::move( vec );

	LOG( view.size(), view.data() );

	view.emitAndReplace();

	LOG( view.size(), view.data() );

	PolyVectorView<int> nn( view.size() );
	PolyVectorView<const int *> pp( view.size() );

	LOG( nn.space(), nn.data() );
	LOG( pp.space(), pp.data() );

	LOG( "size of nn", nn.size() );

	nn.emitAndReplace();
	pp.emitAndReplace();

	LOG( nn.space(), nn.data() );
	LOG( pp.space(), pp.data() );

	LOG( "size of nn", nn.size() );

	kernel( add, 1, 1 )( view, nn, pp );

	LOG( nn.space(), nn.data() );
	LOG( pp.space(), pp.data() );

	LOG( "size of nn", nn.size() );

	nn.fetchAndReplace();
	pp.fetchAndReplace();

	LOG( nn.space(), nn.data() );
	LOG( pp.space(), pp.data() );

	LOG( "size of nn", nn.size() );

	for ( auto &e : nn )
		std::cout << e << std::endl;

	LOG( "normal exit" );
#else
	LOG( "no cuda toolkit provided" );

	EXPECT_EQ( 1, 1 );
#endif
}