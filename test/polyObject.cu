#include "hip/hip_runtime.h"
#include <core/basic/poly.hpp>
#include <gtest/gtest.h>

using namespace koishi;
using namespace core;

TEST( test_poly_object, object_create )
{
	poly::object<int> a = poly::make_object<int>( 1 );
	ASSERT_EQ( *a, 1 );
	*a = 2;
	ASSERT_EQ( *a, 2 );
}

struct A : emittable<A>
{
	A() { n++; }
	A( const A & ) { n++; }
	A( A && ) { 
		printf("move A\n");
#ifndef __CUDA_ARCH__
		n++;
#endif
	}
	A &operator=( A && ) = default;
	A &operator=( const A & ) = default;
	~A() { n--; }
	
	KOISHI_HOST_DEVICE virtual int f() const
	{
		return 1;
	}

	int x = 1;
	static int n;
};

struct B : emittable<B, A>
{
	B() { n++; }
	B( const B & ) { n++; }
	B( B && ) { 
		printf("move B\n");
#ifndef __CUDA_ARCH__
		n++;
#endif
	}
	B &operator=( B && ) = default;
	B &operator=( const B & ) = default;
	~B() { n--; }

	KOISHI_HOST_DEVICE int f() const override
	{
		return 2;
	}

	int y = 2;
	static int n;
};

int A::n = 0;
int B::n = 0;

TEST( test_poly_object, object_sealed )
{
	ASSERT_EQ( A::n, 0 );
	ASSERT_EQ( B::n, 0 );
	{
		poly::object<A> a = poly::make_object<A>();
		ASSERT_EQ( A::n, 1 );
		ASSERT_EQ( B::n, 0 );
		ASSERT_EQ( a->x, 1 );
		{
			poly::object<A> b = poly::make_object<B>();
			ASSERT_EQ( A::n, 2 );
			ASSERT_EQ( B::n, 1 );
			ASSERT_EQ( b->x, 1 );
			// ASSERT_EQ( b->y, 2 );
		}
		ASSERT_EQ( A::n, 1 );
		ASSERT_EQ( B::n, 0 );
	}
	ASSERT_EQ( A::n, 0 );
	ASSERT_EQ( B::n, 0 );
}

TEST( test_poly_object, object_sealed_2 )
{
	ASSERT_EQ( A::n, 0 );
	ASSERT_EQ( B::n, 0 );
	{
		poly::object<A> a = poly::make_object<A>();
		ASSERT_EQ( A::n, 1 );
		ASSERT_EQ( B::n, 0 );
		ASSERT_EQ( a->x, 1 );
		{
			poly::object<A> b = poly::make_object<B>();
			ASSERT_EQ( A::n, 2 );
			ASSERT_EQ( B::n, 1 );
			ASSERT_EQ( b->x, 1 );
			a = std::move( b );
			ASSERT_EQ( A::n, 1 );
			ASSERT_EQ( B::n, 1 );
		}
		ASSERT_EQ( A::n, 1 );
		ASSERT_EQ( B::n, 1 );
	}
	ASSERT_EQ( A::n, 0 );
	ASSERT_EQ( B::n, 0 );
}

TEST( test_poly_object, object_sealed_3 )
{
	ASSERT_EQ( A::n, 0 );
	ASSERT_EQ( B::n, 0 );
	{
		poly::object<A> a = poly::make_object<A>();
		ASSERT_EQ( A::n, 1 );
		ASSERT_EQ( B::n, 0 );
		ASSERT_EQ( a->x, 1 );
		{
			poly::object<A> b = poly::make_object<B>();
			ASSERT_EQ( A::n, 2 );
			ASSERT_EQ( B::n, 1 );
			ASSERT_EQ( b->x, 1 );
			poly::object<B> c = poly::static_object_cast<B>( std::move( b ) );
			ASSERT_EQ( A::n, 2 );
			ASSERT_EQ( B::n, 1 );
			ASSERT_EQ( c->x, 1 );
			ASSERT_EQ( c->y, 2 );
		}
		ASSERT_EQ( A::n, 1 );
		ASSERT_EQ( B::n, 0 );
	}
	ASSERT_EQ( A::n, 0 );
	ASSERT_EQ( B::n, 0 );
}

#ifdef KOISHI_USE_CUDA
__global__ void g( 
	poly::vector<int> &b,
	const poly::object<A> &a
)
{
	// b[ 0 ] = a->x;
	//b[ 1 ] = a->f();
}
#endif

TEST( test_poly_object, object_polymorphism )
{
#ifdef KOISHI_USE_CUDA
	poly::object<A> a = poly::make_object<B>();
	poly::vector<int> b( 2 );
	poly::kernel( g, 1, 1 )( b, a );
	EXPECT_EQ( b[ 0 ], 1 );
	EXPECT_EQ( b[ 1 ], 2 );
#endif
}
